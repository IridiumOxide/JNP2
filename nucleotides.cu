#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include<iostream>
#include<vector>
#include<string>
#include<algorithm>
#include<iomanip>

#include<thrust/device_vector.h>
#include<thrust/host_vector.h>

const int TGM_VALUE_BASE = 5;
const int TGM_VALUE_CB = TGM_VALUE_BASE * TGM_VALUE_BASE * TGM_VALUE_BASE;
const int NEIGHBORHOOD_RADIUS = 128;
const int MINIMUM_WEIGHT = 10;
const int BIG_NUMBER = 2000000000;
const int STRAND_LENGTH = 60;
const int GAP_PENALTY = -2;
const int BLOCKS_PER_INVOCATION = 16;


/*******************
 *  HOST FUNCTIONS *
 *******************/


int nucleotide_value(char x);

class Strand{
private:
    bool complementary;
    long int id;
    std::string original_form;

    std::vector<int> key;
    static int trigram_counts[TGM_VALUE_CB];

    static bool trigram_comparator(const int& a, const int& b) {
        if (trigram_counts[a] == trigram_counts[b]){
            return a < b;
        }
        else{
            return trigram_counts[a] > trigram_counts[b];
        }
    }


public:
    Strand() :
        id(0),
        original_form(""),
        complementary(false)
    {
    }


    Strand(int id, const std::string& original_form, bool complementary) :
        id(id),
        original_form(original_form),
        complementary(complementary)
    {
        // create the key by parsing and sorting nucleotides

        memset(trigram_counts, 0, sizeof(trigram_counts));
        const int tgm_sq = TGM_VALUE_BASE * TGM_VALUE_BASE;
        int current_trigram_value = nucleotide_value(original_form[0]) * TGM_VALUE_BASE +
            nucleotide_value(original_form[1]);
        for (unsigned int i = 2; i < original_form.length(); ++i){
            current_trigram_value %= tgm_sq;
            current_trigram_value *= TGM_VALUE_BASE;
            current_trigram_value += nucleotide_value(original_form[i]);
            trigram_counts[current_trigram_value]++;
            key.push_back(current_trigram_value);
        }
        std::sort(key.begin(), key.end(), trigram_comparator);
    }


    bool operator< (const Strand& other) const{
        for (unsigned int i = 0; i < key.size(); ++i){
            if (i == other.key.size()){
                // other's key is a substring of our key
                return false;
            }
            if (key[i] != other.key[i]){
                return key[i] < other.key[i];
            }
        }
        if (key.size() == other.key.size()){
            return false;
        }
        // end of key; our key is a substring of other's key
        return true;
    }

    std::string& get_original_form(){
        return original_form;
    }

    int get_id() const{
        return id;
    }

    bool is_complementary() const{
        return complementary;
    }
};

int Strand::trigram_counts[TGM_VALUE_CB];

struct ResultEdge{
    int weight;
};


int nucleotide_value(char x){
    switch (x){
    case 'a':
        return 0;
    case 'c':
        return 1;
    case 'g':
        return 2;
    case 't':
        return 3;
    default:
        return 4;
    }
}


std::string complementary(std::string strand){
    std::string result = "";
    for (unsigned int i = 0; i < strand.length(); ++i){
        switch (strand[i]){
        case 'a':
            result += 't';
            break;
        case 'c':
            result += 'g';
            break;
        case 'g':
            result += 'c';
            break;
        case 't':
            result += 'a';
            break;
        default:
            result += strand[i];
            break;
        }
    }
    // return reversed result
    return std::string(result.rbegin(), result.rend());
}


// get a vector of DNA fragments from standard input
void parse_input(std::vector<Strand>& strands){
    const char INFO_LINE_SYMBOL = '>';
    std::string line;
    std::string id_string;

    long int id = 0;
    std::string strand = "";

    while (std::getline(std::cin, line)){
        if (line[0] == INFO_LINE_SYMBOL){
            // previous strand finished; push it into the vector
            if (strand != ""){
                strands.push_back(Strand(id, strand, false));
                strands.push_back(Strand(id, complementary(strand), true));
                if (id % 100 == 0){
                    std::cerr << "Parsed: " << std::setfill('0') << std::setw(7) << id << std::endl;
                }
            }
            strand = "";

            // get id - it starts at line[3]
            id_string = "";
            for (unsigned int i = 3; i < line.length(); ++i){
                id_string += line[i];
            }
            id = std::atoi(id_string.c_str());
        }
        else{
            // this line contains fragment of a strand.
            strand += line;
        }
    }
    // last strand is finished - we should push it into the vector, too.
    strands.push_back(Strand(id, strand, false));
    strands.push_back(Strand(id, complementary(strand), true));
    std::cerr << "Parsed: " << std::setfill('0') << std::setw(7) << id << std::endl;
}

/********************
*  DEVICE FUNCTIONS *
*********************/

struct SimpleStrand{
    char nucleotides[STRAND_LENGTH];
};

__device__ int device_max(int a, int b){
    if (a > b){
        return a;
    }
    return b;
}

__global__ void calculate_values(int starting_n, SimpleStrand *sstrands, int n_strands, ResultEdge *resultEdges){

    int first_strand_id = blockIdx.x + starting_n;
    if (first_strand_id >= n_strands){
        resultEdges[blockIdx.x * NEIGHBORHOOD_RADIUS * 2 + threadIdx.x].weight = -1;
        return;
    }
    int start = first_strand_id - NEIGHBORHOOD_RADIUS;
    start -= device_max(0, NEIGHBORHOOD_RADIUS - n_strands + first_strand_id + 1);
    start = device_max(0, start);
    int second_strand_id = start + threadIdx.x;

    bool should_calculate = false;
    resultEdges[blockIdx.x * NEIGHBORHOOD_RADIUS * 2 + threadIdx.x].weight = -1;

    if (second_strand_id < first_strand_id){
        // we shouldn't compare two same strands twice
        int second_start = device_max(0, second_strand_id - NEIGHBORHOOD_RADIUS);
        int second_end = second_start + 2 * NEIGHBORHOOD_RADIUS;
        should_calculate = (second_end < first_strand_id);
    }
    else{
        second_strand_id += 1;
        should_calculate = (second_strand_id < n_strands);
    }

    if (should_calculate){
        SimpleStrand *first = sstrands + first_strand_id;
        SimpleStrand *second = sstrands + second_strand_id;

        // printf("%d %d\n-1- %c %c --\n-2- %c %c --\n", first_strand_id, second_strand_id, first->nucleotides[0], first->nucleotides[1], second->nucleotides[0], second->nucleotides[1]);

        int best_value = -BIG_NUMBER;
        int matrix_row[2][STRAND_LENGTH];
        int current_row = 0;
        int prev_row = 1;
        // fill first row
        for (int i = 0; i < STRAND_LENGTH; ++i){
            matrix_row[current_row][i] = 0;
        }
        for (int i = 1; i <= STRAND_LENGTH; ++i){
            prev_row = current_row;
            current_row += 1;
            current_row %= 2;
            matrix_row[current_row][0] = 0;
            for (int j = 1; j <= STRAND_LENGTH; ++j){
                if (first->nucleotides[i - 1] == second->nucleotides[j - 1]){
                    matrix_row[current_row][j] = 1;
                }
                else{
                    matrix_row[current_row][j] = -2;
                }
                int m = matrix_row[prev_row][j - 1] + matrix_row[current_row][j];
                m = device_max(m, matrix_row[prev_row][j] + GAP_PENALTY);
                m = device_max(m, matrix_row[current_row][j - 1] + GAP_PENALTY);
                matrix_row[current_row][j] = m;
                if (j == STRAND_LENGTH || i == STRAND_LENGTH){
                    if (m > best_value){
                        best_value = m;
                    }
                }
            }
        }
        resultEdges[blockIdx.x * NEIGHBORHOOD_RADIUS * 2 + threadIdx.x].weight = best_value;
    }
}


/****************
 *     MAIN     *
 ****************/


int main(){
    std::ios_base::sync_with_stdio(false);
    int counter = 0;
    std::cerr << "start" << std::endl;

    std::vector<Strand> strands;
    parse_input(strands);
    std::cerr << "parsed" << std::endl;

    std::sort(strands.begin(), strands.end());
    std::cerr << "sorted" << std::endl;

    int n_strands = strands.size();

    // create array of SimpleStrands on device
    thrust::host_vector<SimpleStrand> host_strands;
    thrust::device_vector<SimpleStrand> device_strands;
    SimpleStrand temp;
    for (unsigned int i = 0; i < n_strands; ++i){
        for (unsigned int j = 0; j < STRAND_LENGTH; ++j){
            temp.nucleotides[j] = strands[i].get_original_form()[j];
        }
        host_strands.push_back(temp);
    }
    device_strands = host_strands;
    SimpleStrand *simple_strands = thrust::raw_pointer_cast(device_strands.data());

    ResultEdge *dev_result_edges, *host_result_edges;
    host_result_edges = (ResultEdge*) malloc(BLOCKS_PER_INVOCATION * sizeof(ResultEdge) * 2 * NEIGHBORHOOD_RADIUS);
    hipMalloc(&dev_result_edges, BLOCKS_PER_INVOCATION * sizeof(ResultEdge) * 2 * NEIGHBORHOOD_RADIUS);

    int last_i = 0;

    std::cerr << "Memory ready" << std::endl;

    // invoke kernel
    calculate_values <<< BLOCKS_PER_INVOCATION, 2 * NEIGHBORHOOD_RADIUS >>>(0, simple_strands, n_strands, dev_result_edges);
    hipMemcpy(host_result_edges, dev_result_edges, BLOCKS_PER_INVOCATION * sizeof(ResultEdge) * 2 * NEIGHBORHOOD_RADIUS, hipMemcpyDeviceToHost);
    for (int i = BLOCKS_PER_INVOCATION; i < n_strands; i += BLOCKS_PER_INVOCATION){
        std::cerr << "next loop  (" << std::setfill('0') << std::setw(7) << i << ")" << std::endl;
        // invoke next kernel
        calculate_values <<< BLOCKS_PER_INVOCATION, 2 * NEIGHBORHOOD_RADIUS >>>(i, simple_strands, n_strands, dev_result_edges);

        // output previous data
        for (int j = 0; j < BLOCKS_PER_INVOCATION; ++j){
            int index = j + last_i;
            int base = j * 2 * NEIGHBORHOOD_RADIUS; // where in the host_result_edges buffer does the data begin
            int start = index - NEIGHBORHOOD_RADIUS;
            start -= std::max(0, NEIGHBORHOOD_RADIUS - n_strands + index + 1);
            start = std::max(0, start);
            for (int k = 0; k < NEIGHBORHOOD_RADIUS * 2; ++k){
                int current = start + k;
                if (current >= index){
                    current += 1;
                }
                if (current >= n_strands){
                    break;
                }
                if (host_result_edges[base + k].weight < MINIMUM_WEIGHT){
                    continue;
                }
                counter++;
                std::cout << "FG " << strands[index].get_id() << "; FG " << strands[current].get_id() << "; ";
                std::cout << host_result_edges[base + k].weight << "; " << strands[index].is_complementary() << "; ";
                std::cout << strands[current].is_complementary() << std::endl;
            }

        }

        // sync with device, get next data
        last_i = i;
        hipMemcpy(host_result_edges, dev_result_edges, BLOCKS_PER_INVOCATION * sizeof(ResultEdge) * 2 * NEIGHBORHOOD_RADIUS, hipMemcpyDeviceToHost);
    }
    // output last batch of data
    for (int j = 0; j < BLOCKS_PER_INVOCATION; ++j){
        int index = j + last_i - BLOCKS_PER_INVOCATION;
        int base = j * 2 * NEIGHBORHOOD_RADIUS;
        int start = index - NEIGHBORHOOD_RADIUS;
        start -= std::max(0, NEIGHBORHOOD_RADIUS - n_strands + index + 1);
        start = std::max(0, start);
        for (int k = 0; k < NEIGHBORHOOD_RADIUS * 2; ++k){
            int current = start + k;
            if (current >= index){
                current += 1;
            }
            if (current >= n_strands){
                break;
            }
            if (host_result_edges[base + k].weight < MINIMUM_WEIGHT){
                continue;
            }
            counter++;
            std::cout << "FG " << strands[index].get_id() << "; FG " << strands[current].get_id() << "; ";
            std::cout << host_result_edges[base + k].weight << "; " << strands[index].is_complementary() << "; ";
            std::cout << strands[current].is_complementary() << std::endl;
        }

    }
    std::cerr << counter << std::endl;
    return 0;
}